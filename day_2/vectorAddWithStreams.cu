// #include <stdio.h>
// #include "../day_common/day_common.h"
// #include <cuda_runtime.h>

// /*
// An example of a vector addition on the host and device. Using a simple kernel to add two
// vectors together. The kernel is launched with a grid of blocks, where each block contains a
// number of threads. Each thread is responsible for adding one element of the two vectors
// together.
// */

// void initializeData(float *ip, int size)
// {
//     // generate different seed for random number
//     time_t t;
//     srand((unsigned)time(&t));
//     for (int i = 0; i < size; i++)
//     {
//         ip[i] = (float)(rand() & 0xFF) / 10.0f;
//     }
// }

// void vectorAddOnHost(float *A, float *B, float *C, const int N)
// {
//     for (int i = 0; i < N; i++)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

// void checkResult(float *hostRef, float *gpuRef, const int N)
// {
//     double epsilon = 1.0E-8;
//     bool match = 1;
//     for (int i = 0; i < N; i++)
//     {
//         if (abs(hostRef[i] - gpuRef[i]) > epsilon)
//         {
//             match = 0;
//             printf("Arrays do not match!\n");
//             printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
//             break;
//         }
//     }
//     if (match)
//         printf("Arrays match.\n");
// }

// __global__ void vectorAddOnDevice(float *A, float *B, float *C, const int N)
// {
//     // Adding the keyword __global__ means that the CPU host will launch this
//     // kernel on the device.
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < N)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

// int main(void)
// {
//     printf("Starting... \n");

//     // This is no limit to the number of streams
//     // but the hardware can only support a number a
//     // certain number of concurrent kernel.
//     int numStreams = 2;

//     // Setting size of my arrays N = 1M elements
//     const int N = 1 << 20;
//     size_t vectorSizeBytes = N * sizeof(float);
//     // number of threads per block
//     int threadsPerBlock = 256;

//     // Calculate base chunk size (integer division)
//     // Handle cases where N might be smaller than numStreams
//     int baseChunkSize = (N > 0) ? (N + numStreams - 1) / numStreams : 0;
//     printf("Vector size: %d that will be of size %d bytes (sizeof(float)=%d) \n", N, vectorSizeBytes, sizeof(float));
//     printf("Base chunk size (approx): %d elements\n", baseChunkSize);

//     float *h_A, *h_B, *h_C, *h_C_prime;
//     cudaMallocHost((void **)&h_A, vectorSizeBytes);
//     cudaMallocHost((void **)&h_B, vectorSizeBytes);
//     cudaMallocHost((void **)&h_C, vectorSizeBytes);
//     cudaMallocHost((void **)&h_C_prime, vectorSizeBytes);

//     // Initialize the arrays on the host
//     double iStart = seconds();
//     initializeData(h_A, N);
//     initializeData(h_B, N);
//     double iElaps = seconds() - iStart;
//     printf("Init time for vectors: %.5f sec\n", iElaps);

//     // memset comes from the std C library
//     memset(h_C, 0, vectorSizeBytes);
//     memset(h_C_prime, 0, vectorSizeBytes);

//     // Vector addition on the host
//     double iStart2 = seconds();
//     vectorAddOnHost(h_A, h_B, h_C, N);
//     double iElaps2 = seconds() - iStart2;
//     printf("Vector addition on the host: %.5f sec\n", iElaps2);

//     // Allocate device memory
//     float *d_A, *d_B, *d_C;
//     cudaMalloc((float **)&d_A, vectorSizeBytes);
//     cudaMalloc((float **)&d_B, vectorSizeBytes);
//     cudaMalloc((float **)&d_C, vectorSizeBytes);

//     // Allocate and initialize an array of stream handles
//     cudaStream_t *streams = (cudaStream_t *)malloc(numStreams * sizeof(
//                                                                     cudaStream_t));

//     for (int i = 0; i < numStreams; i++)
//     {
//         cudaStreamCreate(&(streams[i]));
//     }

//     // --- Timing ---
//     cudaEvent_t start, stop;
//     cudaEventCreate(&start);
//     cudaEventCreate(&stop);

//     // --- Execute with Streams ---
//     printf("Starting GPU vector add with streams...\n");
//     cudaEventRecord(start); // Record start time

//     int currentOffset = 0; // Keep track of the starting position for the next chunk
//     for (int i = 0; i < numStreams; ++i)
//     {
//         // Calculate the number of elements for THIS stream's chunk
//         // Make sure we don't go past the end of the vector N
//         int elementsInThisChunk = std::min(baseChunkSize, N - currentOffset);

//         // If elementsInThisChunk is 0 or negative, it means we've processed all elements
//         // in previous streams (this can happen if n < numStreams). Break the loop.
//         if (elementsInThisChunk <= 0)
//         {
//             printf("Stream %d has no work (all elements assigned).\n", i);
//             break;
//         }

//         size_t currentChunkSizeBytes = elementsInThisChunk * sizeof(float);
//         printf("Stream %d: offset=%d, elements=%d, bytes=%.2f KB\n",
//                i, currentOffset, elementsInThisChunk, (float)currentChunkSizeBytes / 1024.0f);

//         // 1. Asynchronous Host-to-Device Copy for chunk i
//         cudaMemcpyAsync(d_A + currentOffset, h_A + currentOffset, currentChunkSizeBytes, cudaMemcpyHostToDevice, streams[i]);
//         cudaMemcpyAsync(d_B + currentOffset, h_B + currentOffset, currentChunkSizeBytes, cudaMemcpyHostToDevice, streams[i]);

//         // 2. Launch Kernel for chunk i in its stream
//         int blocksPerGrid = (elementsInThisChunk + threadsPerBlock - 1) / threadsPerBlock;
//         if (blocksPerGrid > 0)
//         { // Only launch kernel if there's work to do
//             vectorAddOnDevice<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
//                 d_A + currentOffset,
//                 d_A + currentOffset,
//                 d_C + currentOffset,
//                 elementsInThisChunk // Pass the actual size for this chunk
//             );
//             // Check for kernel launch errors (asynchronous)
//             cudaError_t kernelError = cudaGetLastError();
//             if (kernelError != cudaSuccess)
//             {
//                 fprintf(stderr, "Kernel launch error in stream %d: %s\n", i, cudaGetErrorString(kernelError));
//             }
//         }
//         else
//         {
//             printf("Stream %d: Skipping kernel launch (0 blocks).\n", i);
//         }

//         // 3. Asynchronous Device-to-Host Copy for chunk i
//         cudaMemcpyAsync(h_C_prime + currentOffset, d_C + currentOffset, currentChunkSizeBytes, cudaMemcpyDeviceToHost, streams[i]);

//         // Update offset for the next stream's chunk
//         currentOffset += elementsInThisChunk;
//     }
//     // Ensure we have processed all elements if the loop finished normally
//     if (currentOffset != N && N > 0)
//     {
//         fprintf(stderr, "Error: Loop finished but not all elements processed. currentOffset=%d, n=%d\n", currentOffset, N);
//         // Handle error appropriately - maybe exit or throw exception
//     }

//     // --- Synchronization ---
//     cudaDeviceSynchronize();

//     // --- Timing Stop ---
//     cudaEventRecord(stop);
//     cudaEventSynchronize(stop);
//     float milliseconds = 0;
//     cudaEventElapsedTime(&milliseconds, start, stop);
//     printf("GPU execution time (streams): %.3f ms\n", milliseconds);
//     if (milliseconds > 0)
//     {
//         double totalBytesTransferred = 3.0 * vectorSizeBytes; // 2 reads H->D, 1 write D->H
//         printf("Effective Bandwidth: %.2f GB/s\n", (totalBytesTransferred / (milliseconds / 1000.0)) / (1024.0 * 1024.0 * 1024.0));
//     }

//     checkResult(h_C, h_C_prime, N);

//     cudaDeviceReset();

//     // Reset device and exit
//     printf("Done\n");
//     return 0;
// }

#include <stdio.h>
#include <stdlib.h>  // For malloc, free, srand, abs
#include <time.h>    // For time()
#include <string.h>  // For memset
#include <math.h>    // For abs with doubles/floats
#include <algorithm> // For std::min
#include <vector>    // Optional: Can be useful but sticking to pointers for now

#include <hip/hip_runtime.h>

// Assuming day_common.h provides the seconds() function. If not, implement it.
// #include "../day_common/day_common.h"

// --- Simple replacement for seconds() if day_common.h isn't available ---
#include <sys/time.h> // For gettimeofday() on Linux/macOS
double seconds()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}
// --- End replacement ---

// Simple CUDA error checking macro
#define CUDA_CHECK(call)                                                                                                 \
    do                                                                                                                   \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (err != hipSuccess)                                                                                          \
        {                                                                                                                \
            fprintf(stderr, "CUDA Error in %s at line %d: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(err), err); \
            /* Attempt to reset device before exiting might help cleanup */                                              \
            hipDeviceReset();                                                                                           \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    } while (0)

void initializeData(float *ip, int size)
{
    // generate different seed for random number
    srand((unsigned)time(NULL)); // Use NULL for current time
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void vectorAddOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = true; // Use true/false
    int mismatch_count = 0;
    const int max_mismatches_to_print = 10;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = false;
            mismatch_count++;
            if (mismatch_count <= max_mismatches_to_print)
            {
                printf("Mismatch at index %d: host=%.5f gpu=%.5f diff=%.5e\n",
                       i, hostRef[i], gpuRef[i], abs(hostRef[i] - gpuRef[i]));
            }
        }
    }

    if (match)
    {
        printf("Arrays match.\n");
    }
    else
    {
        printf("ARRAYS DO NOT MATCH! Total mismatches: %d\n", mismatch_count);
        if (mismatch_count > max_mismatches_to_print)
        {
            printf(" (Only the first %d mismatches were printed)\n", max_mismatches_to_print);
        }
    }
}

__global__ void vectorAddOnDevice(float *A, float *B, float *C, const int N_chunk) // N_chunk is size of this specific chunk
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Important: Boundary check uses N_chunk, not the total N
    if (i < N_chunk)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    printf("Starting Vector Add with CUDA Streams...\n");

    // --- Configuration ---
    int numStreams = 2;        // Experiment with different numbers (e.g., 2, 4, 8)
    const int N = 1 << 20;     // Increase problem size for streams (e.g., 16M elements)
    int threadsPerBlock = 256; // Common choice

    size_t vectorSizeBytes = (size_t)N * sizeof(float); // Use size_t for large sizes
    printf("Configuration:\n");
    printf("Vector elements (N): %d (%.2f Million)\n", N, (float)N / (1 << 20));
    printf("Vector size: %zu bytes (%.2f MB)\n", vectorSizeBytes, (float)vectorSizeBytes / (1 << 20));
    printf("CUDA Streams: %d\n", numStreams);
    printf("Threads per Block: %d\n", threadsPerBlock);

    // --- Host Memory Allocation (Pinned Memory Recommended for Async Transfers) ---
    float *h_A, *h_B, *h_C, *h_C_prime;
    printf("Allocating Pinned Host Memory...\n");
    CUDA_CHECK(hipHostMalloc((void **)&h_A, vectorSizeBytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc((void **)&h_B, vectorSizeBytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc((void **)&h_C, vectorSizeBytes, hipHostMallocDefault));       // For host result
    CUDA_CHECK(hipHostMalloc((void **)&h_C_prime, vectorSizeBytes, hipHostMallocDefault)); // For GPU result copied back
    if (!h_A || !h_B || !h_C || !h_C_prime)
    {
        fprintf(stderr, "Failed to allocate host memory!\n");
        hipDeviceReset();
        return EXIT_FAILURE;
    }

    // --- Initialize Host Data ---
    printf("Initializing host data...\n");
    double iStartInit = seconds();
    initializeData(h_A, N);
    initializeData(h_B, N);
    memset(h_C_prime, 0, vectorSizeBytes); // Zero out GPU result buffer
    double iEndInit = seconds();
    printf("Host data initialization time: %.5f sec\n", iEndInit - iStartInit);

    // --- Host Calculation (for verification) ---
    printf("Performing vector addition on host...\n");
    double iStartHost = seconds();
    vectorAddOnHost(h_A, h_B, h_C, N);
    double iEndHost = seconds();
    printf("Host vector addition time: %.5f sec\n", iEndHost - iStartHost);

    // --- Device Memory Allocation ---
    float *d_A, *d_B, *d_C;
    printf("Allocating Device Memory...\n");
    CUDA_CHECK(hipMalloc((float **)&d_A, vectorSizeBytes));
    CUDA_CHECK(hipMalloc((float **)&d_B, vectorSizeBytes));
    CUDA_CHECK(hipMalloc((float **)&d_C, vectorSizeBytes));

    // --- Create CUDA Streams ---
    printf("Creating %d CUDA streams...\n", numStreams);
    hipStream_t *streams = (hipStream_t *)malloc(numStreams * sizeof(hipStream_t));
    if (!streams)
    {
        fprintf(stderr, "Failed to allocate memory for stream handles!\n");
        // Clean up allocated CUDA resources before exiting
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipHostFree(h_A);
        hipHostFree(h_B);
        hipHostFree(h_C);
        hipHostFree(h_C_prime);
        hipDeviceReset();
        return EXIT_FAILURE;
    }
    for (int i = 0; i < numStreams; i++)
    {
        CUDA_CHECK(hipStreamCreate(&(streams[i])));
    }

    // --- Create CUDA Events for Timing ---
    hipEvent_t start_event, stop_event;
    printf("Creating CUDA events for timing...\n");
    CUDA_CHECK(hipEventCreate(&start_event));
    CUDA_CHECK(hipEventCreate(&stop_event));

    // --- Execute Operations with Streams ---
    printf("\nStarting GPU execution with %d streams...\n", numStreams);

    // <<< Record Start Event >>>
    // Record in the default stream (0) before queueing any work.
    CUDA_CHECK(hipEventRecord(start_event, 0));

    int currentOffset = 0;
    for (int i = 0; i < numStreams; ++i)
    {
        // Calculate the number of elements and size for THIS stream's chunk
        int elementsInThisChunk = N / numStreams; // Integer division gives base size
        int remainder = N % numStreams;
        if (i < remainder)
        { // Distribute remainder among the first few streams
            elementsInThisChunk++;
        }

        if (elementsInThisChunk <= 0)
        {
            // This shouldn't happen with the improved distribution, but good practice
            printf("Stream %d: No elements to process, skipping.\n", i);
            continue; // Skip to the next stream
        }

        size_t currentChunkSizeBytes = (size_t)elementsInThisChunk * sizeof(float);
        printf("  Stream %d: Offset=%d, Elements=%d, Size=%.2f KB\n",
               i, currentOffset, elementsInThisChunk, (float)currentChunkSizeBytes / 1024.0f);

        // Define pointers for the current chunk's start
        float *h_A_chunk = h_A + currentOffset;
        float *h_B_chunk = h_B + currentOffset;
        float *d_A_chunk = d_A + currentOffset;
        float *d_B_chunk = d_B + currentOffset;
        float *d_C_chunk = d_C + currentOffset;
        float *h_C_prime_chunk = h_C_prime + currentOffset;

        // 1. Asynchronous Host-to-Device Copy for chunk i
        CUDA_CHECK(hipMemcpyAsync(d_A_chunk, h_A_chunk, currentChunkSizeBytes, hipMemcpyHostToDevice, streams[i]));
        CUDA_CHECK(hipMemcpyAsync(d_B_chunk, h_B_chunk, currentChunkSizeBytes, hipMemcpyHostToDevice, streams[i]));

        // 2. Launch Kernel for chunk i in its stream
        int blocksPerGrid = (elementsInThisChunk + threadsPerBlock - 1) / threadsPerBlock;
        if (blocksPerGrid > 0)
        {
            vectorAddOnDevice<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
                d_A_chunk,
                d_B_chunk,
                d_C_chunk,
                elementsInThisChunk);
            // Check for kernel launch errors (asynchronous) immediately
            CUDA_CHECK(hipGetLastError());
        }
        else
        {
            printf("  Stream %d: Skipping kernel launch (0 blocks).\n", i);
        }

        // 3. Asynchronous Device-to-Host Copy for chunk i's result
        CUDA_CHECK(hipMemcpyAsync(h_C_prime_chunk, d_C_chunk, currentChunkSizeBytes, hipMemcpyDeviceToHost, streams[i]));

        // Update offset for the next stream's chunk
        currentOffset += elementsInThisChunk;
    }

    // Sanity check: Ensure all elements were processed
    if (currentOffset != N)
    {
        fprintf(stderr, "Error: Loop finished but offset (%d) does not equal N (%d)!\n", currentOffset, N);
        // Handle error - cleanup and exit might be best
    }

    // --- Synchronization ---
    // Wait for all commands in all streams to complete before stopping the timer.
    printf("Synchronizing device...\n");
    CUDA_CHECK(hipDeviceSynchronize());
    printf("Device synchronization complete.\n");

    // --- Timing Stop & Calculation ---
    // <<< Record Stop Event >>>
    // Record in the default stream after all work is done.
    CUDA_CHECK(hipEventRecord(stop_event, 0));

    // Wait for the stop event itself to be processed before calculating time.
    CUDA_CHECK(hipEventSynchronize(stop_event));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));

    printf("\n--- GPU Performance ---\n");
    printf("Total GPU execution time (H2D Copy + Kernel + D2H Copy): %.3f ms\n", milliseconds);

    if (milliseconds > 0)
    {
        // Calculate effective bandwidth: considers H->D for A & B, and D->H for C
        double totalBytesTransferred = 3.0 * vectorSizeBytes;
        double secondsElapsed = milliseconds / 1000.0;
        double gibPerSecond = (totalBytesTransferred / secondsElapsed) / (1024.0 * 1024.0 * 1024.0);
        printf("Effective Memory Bandwidth (Combined H2D + D2H): %.3f GiB/s\n", gibPerSecond);

        // Estimate kernel GFLOPS (N additions = N FLOPs)
        double gflops = (double)N / secondsElapsed / 1e9;
        printf("Kernel Estimated Performance: %.3f GFLOPS\n", gflops);
    }

    // --- Verification ---
    printf("\n--- Verification ---\n");
    checkResult(h_C, h_C_prime, N);

    // --- Cleanup ---
    printf("\nCleaning up resources...\n");
    // Destroy events
    CUDA_CHECK(hipEventDestroy(start_event));
    CUDA_CHECK(hipEventDestroy(stop_event));

    // Destroy streams
    for (int i = 0; i < numStreams; i++)
    {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    free(streams); // Free the array holding stream handles

    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    // Free pinned host memory
    CUDA_CHECK(hipHostFree(h_A));
    CUDA_CHECK(hipHostFree(h_B));
    CUDA_CHECK(hipHostFree(h_C));
    CUDA_CHECK(hipHostFree(h_C_prime));

    // Optional: Reset device, but explicit cleanup is often preferred
    // cudaDeviceReset();

    printf("Done\n");
    return 0;
}