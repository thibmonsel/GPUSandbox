#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string.h>

/*
 * This code implements the neighbor-paired approachs for
 * parallel reduction in CUDA. For this example, the sum operation
 * is used but other operations can be used max, min, average, etc.
 */

#define CUDA_CHECK(call)                                                                                                 \
    do                                                                                                                   \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (err != hipSuccess)                                                                                          \
        {                                                                                                                \
            fprintf(stderr, "CUDA Error in %s at line %d: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(err), err); \
            hipDeviceReset();                                                                                           \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    } while (0)

int recursiveReduce(int *data, int const size)
{
    // terminate check
    if (size == 1)
        return data[0];
    // renew the stride
    int const stride = size / 2;
    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }
    // call recursively
    return recursiveReduce(data, stride);
}

// Neighbored Pair Implementation with "Less" divergence
__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int block_start_idx = blockIdx.x * blockDim.x;

    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        // Instead of `if ((tid % (2*s)) == 0)`, we make the first set of threads active.
        // Each active thread 'tid' will calculate its own target write and source read indices.
        // The 'write_local_idx' corresponds to the 'even' multiples of 's' in the pair.
        // The 'read_local_idx' corresponds to the 'odd' multiples of 's' in the pair.

        unsigned int write_local_idx = tid * 2 * s;
        unsigned int read_local_idx = write_local_idx + s;

        // Condition for this thread to be active:
        // Its calculated read_local_idx must be within the bounds of the block's segment.
        // This effectively makes threads 0 to (blockDim.x / (2*s) - 1) active,
        // leading to contiguous active threads and thus better warp utilization.
        if (read_local_idx < blockDim.x)
        {
            unsigned int write_global_idx = block_start_idx + write_local_idx;
            unsigned int read_global_idx = block_start_idx + read_local_idx;

            // Perform the addition only if BOTH the target write location
            // AND the source read location are within the valid data range [0, n-1].
            if (write_global_idx < n && read_global_idx < n)
            {
                g_idata[write_global_idx] += g_idata[read_global_idx];
            }
        }

        __syncthreads();
    }

    // Thread 0 of each block writes this block's reduced sum to g_odata.
    if (tid == 0)
    {
        if (block_start_idx < n)
        {
            g_odata[blockIdx.x] = g_idata[block_start_idx];
        }
    }
}

void test_reduction()
{
    printf("\n--- Testing Parallel Reduction (Neighbor-Paired) ---\n");

    const int block_size = 256;

    std::vector<size_t> problem_sizes;
    problem_sizes.push_back(1);
    problem_sizes.push_back(block_size / 2);
    problem_sizes.push_back(block_size);
    problem_sizes.push_back(block_size + 1);
    problem_sizes.push_back(1024 * 1024);
    problem_sizes.push_back(1024 * 1024 + 13);
    problem_sizes.push_back(4 * 1024 * 1024);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    for (size_t N : problem_sizes)
    {
        if (N == 0)
        {
            printf("\nSkipping N = 0 (no elements to reduce)\n");
            continue;
        }

        unsigned int num_blocks = (N + block_size - 1) / block_size;
        dim3 gridDim(num_blocks);
        dim3 blockDim(block_size);

        size_t nBytes_input = N * sizeof(int);
        size_t nBytes_output_partial_sums = num_blocks * sizeof(int);

        printf("\nProblem Size N = %zu elements\n", N);
        printf("Configuration: %u blocks, %d threads/block\n", num_blocks, block_size);
        printf("Input data: %.2f MiB, Output (block sums): %.2f KiB\n",
               (double)nBytes_input / (1024.0 * 1024.0),
               (double)nBytes_output_partial_sums / 1024.0);

        std::vector<int> h_idata(N);
        int *d_idata = nullptr;
        int *d_odata = nullptr;

        CUDA_CHECK(hipMalloc(&d_idata, nBytes_input));
        CUDA_CHECK(hipMalloc(&d_odata, nBytes_output_partial_sums));
        CUDA_CHECK(hipMemset(d_odata, 0, nBytes_output_partial_sums));

        long long expected_sum_cpu = 0;
        for (size_t i = 0; i < N; ++i)
        {
            h_idata[i] = (int)((i % 200) - 100);
            expected_sum_cpu += h_idata[i];
        }

        CUDA_CHECK(hipMemcpy(d_idata, h_idata.data(), nBytes_input, hipMemcpyHostToDevice));

        // Warmup run (good practice, especially for timing)
        reduceNeighboredLess<<<gridDim, blockDim>>>(d_idata, d_odata, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(d_idata, h_idata.data(), nBytes_input, hipMemcpyHostToDevice));

        // Timed execution
        CUDA_CHECK(hipEventRecord(start, 0));
        reduceNeighboredLess<<<gridDim, blockDim>>>(d_idata, d_odata, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));

        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

        std::vector<int> h_odata_results(num_blocks);
        CUDA_CHECK(hipMemcpy(h_odata_results.data(), d_odata, nBytes_output_partial_sums, hipMemcpyDeviceToHost));

        long long gpu_final_sum = 0;
        for (unsigned int i = 0; i < num_blocks; ++i)
        {
            gpu_final_sum += h_odata_results[i];
        }

        // Verification
        printf("CPU Sum (expected): %lld\n", expected_sum_cpu);
        printf("GPU Sum (actual):   %lld\n", gpu_final_sum);
        if (expected_sum_cpu == gpu_final_sum)
        {
            printf("Verification: PASS\n");
        }
        else
        {
            printf("Verification: FAIL\n");
            fprintf(stderr, "Error: Mismatch for N = %zu. CPU sum: %lld, GPU sum: %lld\n", N, expected_sum_cpu, gpu_final_sum);
        }

        // Performance Calculation
        double seconds = milliseconds / 1000.0;
        double elements_per_sec = (double)N / seconds;
        double g_elements_per_sec = elements_per_sec / 1e9;

        // Effective Bandwidth Calculation:
        double total_bytes_processed = (double)nBytes_input + (double)nBytes_output_partial_sums;
        double effective_bandwidth_gibps = (total_bytes_processed / seconds) / (1024.0 * 1024.0 * 1024.0); // GiB/s

        printf("Elapsed Time: %.3f ms\n", milliseconds);
        printf("Performance:  %.3f GigaElements/s\n", g_elements_per_sec);
        printf("Effective Bandwidth (kernel part): %.3f GiB/s\n", effective_bandwidth_gibps);

        // Cleanup device memory for this problem size
        CUDA_CHECK(hipFree(d_idata));
        CUDA_CHECK(hipFree(d_odata));
    }

    // Destroy CUDA events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    printf("-----------------------------------------------------\n");
}

int main()
{
    int deviceId;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&deviceId));
    CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));

    printf("Using CUDA Device: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Warp Size: %d\n", prop.warpSize);
    printf("Total Global Memory: %.2f MiB\n", prop.totalGlobalMem / (1024.0 * 1024.0));

    test_reduction();

    CUDA_CHECK(hipDeviceReset());

    printf("\nDone.\n");
    return 0;
}