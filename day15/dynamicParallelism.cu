#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                                                                 \
    do                                                                                                                   \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (err != hipSuccess)                                                                                          \
        {                                                                                                                \
            fprintf(stderr, "CUDA Error in %s at line %d: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(err), err); \
            hipDeviceReset();                                                                                           \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    } while (0)

__global__ void nestedHelloWorld(int const iSize, int iDepth)
{
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d "
           "block %d\n",
           iDepth, tid, blockIdx.x);
    // condition to stop recursive execution
    if (iSize == 1)
        return;
    // reduce block size to half
    int nthreads = iSize >> 1;
    // thread 0 launches child grid recursively
    if (tid == 0 && nthreads > 0)
    {
        nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);
        printf("-------> nested execution depth: %d\n", iDepth);
    }
}

int main(int argc, char **argv)
{
    int size = 8;
    int blocksize = 8; // initial block size
    int igrid = 1;

    if (argc > 1)
    {
        igrid = atoi(argv[1]);
        size = igrid * blocksize;
    }

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("%s Execution Configuration: grid %d block %d\n", argv[0], grid.x,
           block.x);

    nestedHelloWorld<<<grid, block>>>(block.x, 0);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceReset());

    /*
    Looking at the CUDA code example, I can explain why the block ID for the child grids are all 0 in the output messages.
    The reason is in the nested kernel launch inside the nestedHelloWorld function:
    cpp

    nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);

    When a CUDA kernel is launched, the triple angle brackets <<<grid_dim, block_dim>>> specify the execution configuration. The first parameter (1 in this case) defines the number of blocks in the grid. Since it's explicitly set to 1, all child grid launches will have exactly one block, and therefore all child grid executions will report block 0 in their output messages.
    This is why you see all child grids having block ID 0, even though the parent grid might have multiple blocks (depending on the command line arguments passed to the program).
    If you wanted to change this behavior to have multiple blocks in the child grids, you could modify the kernel launch line to something like:

    nestedHelloWorld<<<some_number_of_blocks, nthreads>>>(nthreads, ++iDepth);
    */
    return 0;
}