#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double seconds()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

/*
An example of a vector addition on the host and device. Using a simple kernel to add two
vectors together. The kernel is launched with a grid of blocks, where each block contains a
number of threads. Each thread is responsible for adding one element of the two vectors
together.
*/

#define CUDA_CHECK(call)                                                                                                 \
    do                                                                                                                   \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (err != hipSuccess)                                                                                          \
        {                                                                                                                \
            fprintf(stderr, "CUDA Error in %s at line %d: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(err), err); \
            /* Attempt to reset device before exiting might help cleanup */                                              \
            hipDeviceReset();                                                                                           \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    } while (0)

void initializeData(float *ip, int size)
{
    srand((unsigned)time(NULL));
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void vectorAddOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = true;
    int mismatch_count = 0;
    const int max_mismatches_to_print = 10;

    for (int i = 0; i < N; i++)
    {
        if (fabs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = false;
            mismatch_count++;
            if (mismatch_count <= max_mismatches_to_print)
            {
                printf("Mismatch at index %d: host=%.5f gpu=%.5f diff=%.5e\n",
                       i, hostRef[i], gpuRef[i], fabs(hostRef[i] - gpuRef[i]));
            }
        }
    }

    if (match)
    {
        printf("Arrays match.\n");
    }
    else
    {
        printf("ARRAYS DO NOT MATCH! Total mismatches: %d\n", mismatch_count);
        if (mismatch_count > max_mismatches_to_print)
        {
            printf(" (Only the first %d mismatches were printed)\n", max_mismatches_to_print);
        }
    }
}

__global__ void vectorAddOnDevice(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    printf("Starting Simple Vector Add (Single Stream) - Timing Total GPU Work...\n");

    // --- Configuration ---
    const int N = 1 << 20;
    size_t nBytes = (size_t)N * sizeof(float);
    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    printf("Configuration:\n");
    printf("Vector elements (N): %d (%.2f Million)\n", N, (float)N / (1 << 20));
    printf("Vector size: %zu bytes (%.2f MB)\n", nBytes, (float)nBytes / (1 << 20));
    printf("Kernel launch: %d blocks, %d threads per block\n", numBlocks, numThreadsPerBlock);

    // --- Host Memory Allocation ---
    float *h_A, *h_B, *h_C, *h_C_prime;
    printf("Allocating host memory (%zu bytes)...\n", 4 * nBytes);
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);
    h_C_prime = (float *)malloc(nBytes);
    if (!h_A || !h_B || !h_C || !h_C_prime)
    {
        fprintf(stderr, "Failed to allocate host memory!\n");
        return EXIT_FAILURE;
    }

    // --- Initialize Host Data ---
    printf("Initializing host data...\n");
    double iStartInit = seconds();
    initializeData(h_A, N);
    initializeData(h_B, N);
    memset(h_C_prime, 0, nBytes);
    double iEndInit = seconds();
    printf("Host data initialization time: %.5f sec\n", iEndInit - iStartInit);

    // --- Host Calculation (for verification) ---
    printf("Performing vector addition on host...\n");
    double iStartHost = seconds();
    vectorAddOnHost(h_A, h_B, h_C, N);
    double iEndHost = seconds();
    printf("Host vector addition time: %.5f sec\n", iEndHost - iStartHost);

    // --- Device Memory Allocation ---
    float *d_A, *d_B, *d_C;
    printf("Allocating Device Memory (%zu bytes)...\n", 3 * nBytes);
    CUDA_CHECK(hipMalloc((float **)&d_A, nBytes));
    CUDA_CHECK(hipMalloc((float **)&d_B, nBytes));
    CUDA_CHECK(hipMalloc((float **)&d_C, nBytes));

    // --- GPU Work & Timing (H2D + Kernel + D2H) ---
    hipEvent_t start_gpu_total, stop_gpu_total;
    printf("\nCreating CUDA events for overall GPU timing...\n");
    CUDA_CHECK(hipEventCreate(&start_gpu_total));
    CUDA_CHECK(hipEventCreate(&stop_gpu_total));

    printf("Starting GPU operations and recording events...\n");

    // <<< Record Start Event >>> (Before *first* GPU operation)
    CUDA_CHECK(hipEventRecord(start_gpu_total, 0)); // 0 for default stream

    // 1. H2D Data Transfer
    printf("  Copying data from host to device (A)...\n");
    CUDA_CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    printf("  Copying data from host to device (B)...\n");
    CUDA_CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // 2. GPU Kernel Execution
    printf("  Launching kernel...\n");
    vectorAddOnDevice<<<numBlocks, numThreadsPerBlock>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());

    // 3. D2H Data Transfer
    printf("  Copying result from device to host...\n");
    CUDA_CHECK(hipMemcpy(h_C_prime, d_C, nBytes, hipMemcpyDeviceToHost));

    // <<< Record Stop Event >>> (After *last* GPU operation)
    CUDA_CHECK(hipEventRecord(stop_gpu_total, 0));

    // <<< Synchronize on Stop Event >>>
    // Wait for the GPU to finish executing everything up to the stop event marker
    printf("Waiting for all GPU operations to complete (synchronizing on event)...\n");
    CUDA_CHECK(hipEventSynchronize(stop_gpu_total));

    // <<< Calculate Elapsed Time >>>
    float millisecondsTotal = 0;
    CUDA_CHECK(hipEventElapsedTime(&millisecondsTotal, start_gpu_total, stop_gpu_total));

    printf("\n--- Total GPU Performance ---\n");
    printf("Total GPU execution time (H2D + Kernel + D2H): %.3f ms\n", millisecondsTotal);

    // Estimate Effective Bandwidth & GFLOPS based on total time
    if (millisecondsTotal > 0)
    {
        // Calculate effective bandwidth: considers H->D for A & B, and D->H for C
        double totalBytesTransferred = 3.0 * nBytes;
        double secondsElapsed = millisecondsTotal / 1000.0;
        double gibPerSecond = (totalBytesTransferred / secondsElapsed) / (1024.0 * 1024.0 * 1024.0);
        printf("Effective Memory Bandwidth (Combined H2D + Kernel + D2H): %.3f GiB/s\n", gibPerSecond);

        // GFLOPS calculation remains the same (N additions), but the time base is different
        double gflops = (double)N / secondsElapsed / 1e9;
        printf("Overall Performance including transfers: %.3f GFLOPS\n", gflops);
    }

    // Clean up events immediately after use
    CUDA_CHECK(hipEventDestroy(start_gpu_total));
    CUDA_CHECK(hipEventDestroy(stop_gpu_total));
    // --- End GPU Work & Timing ---

    // --- Verification ---
    printf("\n--- Verification ---\n");
    checkResult(h_C, h_C_prime, N);

    // --- Cleanup ---
    printf("\nCleaning up resources...\n");
    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_prime);

    // Optional: Reset device, but explicit cleanup is often preferred
    // cudaDeviceReset();

    printf("Done\n");
    return 0;
}