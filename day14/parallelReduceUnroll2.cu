#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string.h>

/*
 * This code implements the interleaved-paired approachs for
 * parallel reduction in CUDA with some unrolling of loops.
 */

#define CUDA_CHECK(call)                                                                                                 \
    do                                                                                                                   \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (err != hipSuccess)                                                                                          \
        {                                                                                                                \
            fprintf(stderr, "CUDA Error in %s at line %d: %s (%d)\n", __FILE__, __LINE__, hipGetErrorString(err), err); \
            hipDeviceReset();                                                                                           \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    } while (0)

int recursiveReduce(int *data, int const size)
{
    // terminate check
    if (size == 1)
        return data[0];
    // renew the stride
    int const stride = size / 2;
    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }
    // call recursively
    return recursiveReduce(data, stride);
}

// Interleaved Pair Implementation with warp and data block unrolling
__global__ void reduceInterleavedUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int block_start_idx = blockIdx.x * blockDim.x * 2;

    int *idata = g_idata + block_start_idx;

    // unroll 2 data blocks
    if (block_start_idx + tid + blockDim.x < n)
    {
        g_idata[block_start_idx + tid] += g_idata[block_start_idx + tid + blockDim.x];
    }
    __syncthreads();

    // In-place reduction in global memory using a decreasing stride.
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        // The first 'stride' threads are active.
        if (tid < stride)
        {
            // We need to map the active threads [0, stride-1] to the correct
            // interleaved indices from the previous step.
            // The 'multiplier' determines how far apart the elements are.
            // Ex: stride=256 -> multiplier=2. We add elements at (0,1), (2,3), ...
            // Ex: stride=128 -> multiplier=4. We add elements at (0,2), (4,6), ...
            unsigned int multiplier = (blockDim.x / stride);

            unsigned int write_local_idx = tid * multiplier;
            unsigned int read_local_idx = write_local_idx + (multiplier / 2);

            // We only need to check the read_idx, as the write_idx is guaranteed to be smaller.
            if (block_start_idx + read_local_idx < n)
            {
                idata[write_local_idx] += idata[read_local_idx];
            }
        }

        __syncthreads();
    }

    if (tid < 32)
    {
        int my_sum = 0;
        if (blockDim.x > 32)
        {
            // If the loop ran, the 64 partial sums are interleaved in global memory.
            // We must read from two interleaved locations.
            unsigned int multiplier = blockDim.x / 32;
            unsigned int read_idx1 = tid * multiplier;
            unsigned int read_idx2 = read_idx1 + (multiplier / 2);

            // Check bounds before reading from global memory
            if (block_start_idx + read_idx1 < n)
                my_sum = idata[read_idx1];
            if (block_start_idx + read_idx2 < n)
                my_sum += idata[read_idx2];
        }
        else
        {
            // If the block size was 32 or less, the loop was skipped.
            // Data is at its original position in global memory.
            if (block_start_idx + tid < n)
                my_sum = idata[tid];
        }

        // This full sequence correctly sums the 32 values in `my_sum`.
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, 16);
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, 8);
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, 4);
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, 2);
        my_sum += __shfl_down_sync(0xFFFFFFFF, my_sum, 1);

        // Thread 0 writes the final result.
        if (tid == 0)
        {
            g_odata[blockIdx.x] = my_sum;
        }
    }
}

void test_reduction()
{
    printf("\n--- Testing Parallel Reduction (Interleaved) ---\n");

    const int block_size = 256;

    std::vector<size_t> problem_sizes;
    problem_sizes.push_back(1);
    problem_sizes.push_back(block_size / 2);
    problem_sizes.push_back(block_size);
    problem_sizes.push_back(block_size + 1);
    problem_sizes.push_back(1024 * 1024);
    problem_sizes.push_back(1024 * 1024 + 13);
    problem_sizes.push_back(4 * 1024 * 1024);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    for (size_t N : problem_sizes)
    {
        if (N == 0)
        {
            printf("\nSkipping N = 0 (no elements to reduce)\n");
            continue;
        }

        unsigned int num_blocks = (N + block_size - 1) / block_size;
        dim3 gridDim(num_blocks);
        dim3 blockDim(block_size);

        size_t nBytes_input = N * sizeof(int);
        size_t nBytes_output_partial_sums = num_blocks * sizeof(int);

        printf("\nProblem Size N = %zu elements\n", N);
        printf("Configuration: %u blocks, %d threads/block\n", num_blocks, block_size);
        printf("Input data: %.2f MiB, Output (block sums): %.2f KiB\n",
               (double)nBytes_input / (1024.0 * 1024.0),
               (double)nBytes_output_partial_sums / 1024.0);

        std::vector<int> h_idata(N);
        int *d_idata = nullptr;
        int *d_odata = nullptr;

        CUDA_CHECK(hipMalloc(&d_idata, nBytes_input));
        CUDA_CHECK(hipMalloc(&d_odata, nBytes_output_partial_sums));
        CUDA_CHECK(hipMemset(d_odata, 0, nBytes_output_partial_sums));

        long long expected_sum_cpu = 0;
        for (size_t i = 0; i < N; ++i)
        {
            h_idata[i] = (int)((i % 200) - 100);
            expected_sum_cpu += h_idata[i];
        }

        CUDA_CHECK(hipMemcpy(d_idata, h_idata.data(), nBytes_input, hipMemcpyHostToDevice));

        // Warmup run (good practice, especially for timing)
        reduceInterleavedUnroll<<<gridDim, blockDim>>>(d_idata, d_odata, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(d_idata, h_idata.data(), nBytes_input, hipMemcpyHostToDevice));

        // Timed execution
        CUDA_CHECK(hipEventRecord(start, 0));
        reduceInterleavedUnroll<<<gridDim, blockDim>>>(d_idata, d_odata, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));

        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

        std::vector<int> h_odata_results(num_blocks);
        CUDA_CHECK(hipMemcpy(h_odata_results.data(), d_odata, nBytes_output_partial_sums, hipMemcpyDeviceToHost));

        long long gpu_final_sum = 0;
        for (unsigned int i = 0; i < num_blocks; ++i)
        {
            gpu_final_sum += h_odata_results[i];
        }

        // Verification
        printf("CPU Sum (expected): %lld\n", expected_sum_cpu);
        printf("GPU Sum (actual):   %lld\n", gpu_final_sum);
        if (expected_sum_cpu == gpu_final_sum)
        {
            printf("Verification: PASS\n");
        }
        else
        {
            printf("Verification: FAIL\n");
            fprintf(stderr, "Error: Mismatch for N = %zu. CPU sum: %lld, GPU sum: %lld\n", N, expected_sum_cpu, gpu_final_sum);
        }

        // Performance Calculation
        double seconds = milliseconds / 1000.0;
        double elements_per_sec = (double)N / seconds;
        double g_elements_per_sec = elements_per_sec / 1e9;

        // Effective Bandwidth Calculation:
        double total_bytes_processed = (double)nBytes_input + (double)nBytes_output_partial_sums;
        double effective_bandwidth_gibps = (total_bytes_processed / seconds) / (1024.0 * 1024.0 * 1024.0); // GiB/s

        printf("Elapsed Time: %.3f ms\n", milliseconds);
        printf("Performance:  %.3f GigaElements/s\n", g_elements_per_sec);
        printf("Effective Bandwidth (kernel part): %.3f GiB/s\n", effective_bandwidth_gibps);

        // Cleanup device memory for this problem size
        CUDA_CHECK(hipFree(d_idata));
        CUDA_CHECK(hipFree(d_odata));
    }

    // Destroy CUDA events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    printf("-----------------------------------------------------\n");
}

int main()
{
    int deviceId;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&deviceId));
    CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));

    printf("Using CUDA Device: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Warp Size: %d\n", prop.warpSize);
    printf("Total Global Memory: %.2f MiB\n", prop.totalGlobalMem / (1024.0 * 1024.0));

    printf("Unrooling 2 data blocks in the kernel.\n");
    test_reduction();

    CUDA_CHECK(hipDeviceReset());

    printf("\nDone.\n");
    return 0;
}